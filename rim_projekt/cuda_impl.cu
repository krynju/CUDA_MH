#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <chrono>

#include <stdio.h>
#include <random>
#include <tuple>
#include <math.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hiprand/hiprand_kernel.h>

#define SAMPLES_PER_JOB 10000
#define BLOCK_SIZE 250


// CPU CODE FOR STD TUNING 

float cpu_norm_pdf(float mean, float std, float x) {
    const float a = 1 / sqrt(2 * M_PI);
    return a / std * exp(-0.5 * pow(((x - mean) / std), 2));
}

float cpu_ff(float x) {
    x = x / 100.0;
    return cpu_norm_pdf(0.0, 1.0, x) + cpu_norm_pdf(3.0, 1.0, x) + cpu_norm_pdf(6.0, 1.0, x);
}

std::tuple<float, float> cpu_burn_loop(float xt, float std, int burn_N, float (*f)(float), std::mt19937 gen) {
    float f_xt = f(xt);
    std::uniform_real<float> uni(0, 1);
    const float target = 0.3;
    int accepted = 0;
    for (int t = 0; t < burn_N; t++) {
        std::normal_distribution<> g(xt, std);
        float xc = g(gen);
        float f_xc = f(xc);
        float a = f_xc / f_xt;
        float u = uni(gen);
        if (u <= a) {
            xt = xc;
            f_xt = f_xc;
            accepted += 1;
        }
        float temp = accepted / (t + 1.0);
        float reg = 1 / std * 1000.0 * (temp - target);
        if (!(reg < 0 && abs(reg) > std)) {
            std += reg;
        }
    }
    return std::tuple<float, float>(xt, std);
}


// DEVICE CODE

__device__ float dev_norm_pdf(float mean, float std, float x) {
    const float inv_sqrt_2pi = 0.3989422804014327;
    return inv_sqrt_2pi / std * exp(-pow((x - mean) / std, 2) / 2.0);
}

__device__ float dev_ff(float x) {
    x = x / 100.0;
    return dev_norm_pdf(0.0, 1.0, x) + dev_norm_pdf(3.0, 1.0, x) + dev_norm_pdf(6.0, 1.0, x);
}

__device__ float(*dev_ff_p)(float) = dev_ff;


__global__ void dev_generate(hiprandStateMtgp32* state, float* x, int x_len, float xt, float std, float (*f)(float)) {
    int i = SAMPLES_PER_JOB * BLOCK_SIZE * blockIdx.x   + threadIdx.x * SAMPLES_PER_JOB;
    int sstart_range = i ;
    int eend_range= i + SAMPLES_PER_JOB;

    if (eend_range > x_len) { eend_range = x_len; }
  
    float f_xt = f(xt);
    for (int t = sstart_range; t < eend_range; t++) {
        float xc = xt + hiprand_normal(&state[blockIdx.x])*std ;
        float f_xc = f(xc);
        float a = f_xc / f_xt;
        float u = hiprand_uniform(&state[blockIdx.x]);
  
        if (u <= a) {
            xt = xc;
            f_xt = f_xc;
        }
        x[t] = xt;
    }
}

int mh(float*x,float x0, int N, int burn_N, float (*cpu_f)(float), float (*f)(float), int seed) {
    const int samples_per_block = (BLOCK_SIZE * SAMPLES_PER_JOB);
    const int block_num = (N + samples_per_block - 1) / samples_per_block;

    typedef std::chrono::high_resolution_clock Clock;
    
    hipEvent_t start, stop;
    float elapsedTime;
    Clock::time_point t0 = Clock::now();

    checkCudaErrors(hipSetDevice(0));

    float* dev_x;

    checkCudaErrors(hipMalloc((void**)&dev_x, N * sizeof(float)));

    
    float xt = x0;
    float std = 1.0;

    std::mt19937 gen(seed);

    std::tuple<float, float> res;
    res = cpu_burn_loop( xt, std, burn_N, cpu_f, gen);
    //printf("std = %f \n", std::get<1>(res));

    xt = std::get<0>(res);
    std = std::get<1>(res);

    hiprandStateMtgp32* devMTGPStates;
    mtgp32_kernel_params* devKernelParams;

    checkCudaErrors(hipMalloc((void**)&devMTGPStates, block_num * sizeof(hiprandStateMtgp32)));

    checkCudaErrors(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));

    hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);

    /* Initialize one state per thread block */
    hiprandMakeMTGP32KernelState(devMTGPStates,mtgp32dc_params_fast_11213, devKernelParams, block_num, seed * 117);

    float (*fff)(float);
    checkCudaErrors(hipMemcpyFromSymbol(&fff, HIP_SYMBOL(dev_ff_p), sizeof(f))); //tu nie mogę podać f po prostu tylko muszę ff_gpu_stat

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
   
    dev_generate<<<block_num,BLOCK_SIZE>>>(devMTGPStates, dev_x, N, xt, std, fff);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));


    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(x, dev_x, N * sizeof(float), hipMemcpyDeviceToHost));
  

    Clock::time_point t1 = Clock::now();

    auto d = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0);

    printf("Kernel-only time: %f ms\n", elapsedTime);
    printf("All time %d ms \n", d.count());
 
    checkCudaErrors(hipFree(dev_x));

    return 0;
}

extern "C" float* cuda_main(int N, int burn_N)
{
    float* x = (float*)malloc(sizeof(float) * N);

    mh(x, 0.0, N, burn_N, cpu_ff,dev_ff_p, 1117);

    checkCudaErrors(hipDeviceReset());

    return x;
}

