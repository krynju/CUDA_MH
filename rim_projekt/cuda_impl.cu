#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <chrono>

#include <stdio.h>
#include <random>
#include <tuple>
#include <math.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hiprand/hiprand_kernel.h>

#define SAMPLES_PER_JOB 10000
#define BLOCK_SIZE 250


// CPU CODE FOR STD TUNING 

double cpu_norm_pdf(double mean, float std, float x) {
    const double a = 1 / sqrt(2 * M_PI);
    return a / std * exp(-0.5 * pow(((x - mean) / std), 2));
}

double cpu_ff(float x) {
    x = x / 100.0;
    return double(cpu_norm_pdf(0.0, 1.0, x) + cpu_norm_pdf(3.0, 1.0, x) + cpu_norm_pdf(6.0, 1.0, x));
}

std::tuple<float, float> cpu_burn_loop(float xt, float std, int burn_N, double (*f)(float), std::mt19937 gen) {
    double f_xt = f(xt);
    std::tuple<float, double, double> res;
    std::uniform_real<double> uni(0, 1);
    const float target = 0.3;
    int accepted = 0;
    for (int t = 0; t < burn_N; t++) {
        std::normal_distribution<> g(xt, std);
        float xc = g(gen);
        double f_xc = f(xc);
        double a = f_xc / f_xt;
        float u = uni(gen);
        if (u <= a) {
            xt = xc;
            f_xt = f_xc;
            accepted += 1;
        }
        float temp = accepted / (t + 1.0);
        float reg = 1 / std * 1000.0 * (temp - target);
        if (!(reg < 0 && abs(reg) > std)) {
            std += reg;
        }
    }
    return std::tuple<float, float>(xt, std);
}


// DEVICE CODE

__device__ float dev_norm_pdf(float mean, float std, float x) {
    const float inv_sqrt_2pi = 0.3989422804014327;
    return inv_sqrt_2pi / std * exp(-pow((x - mean) / std, 2) / 2.0);
}

__device__ float dev_ff(float x) {
    x = x / 100.0;
    return dev_norm_pdf(0.0, 1.0, x) + dev_norm_pdf(3.0, 1.0, x) + dev_norm_pdf(6.0, 1.0, x);
}

__device__ float(*dev_ff_p)(float) = dev_ff;


__global__ void _generate_loop_gpu(hiprandStateMtgp32* state, float* x, int x_len, float xt, float std, float (*f)(float)) {
    int i = SAMPLES_PER_JOB * BLOCK_SIZE * blockIdx.x   + threadIdx.x * SAMPLES_PER_JOB;
    int sstart_range = i ;
    int eend_range= i + SAMPLES_PER_JOB;

    if (eend_range > x_len) { eend_range = x_len; }
  
    float f_xt = f(xt);
    for (int t = sstart_range; t < eend_range; t++) {
        float xc = xt + hiprand_normal(&state[blockIdx.x])*std ;
        float f_xc = f(xc);
        float a = f_xc / f_xt;
        float u = hiprand_uniform(&state[blockIdx.x]);
  
        if (u <= a) {
            xt = xc;
            f_xt = f_xc;
        }
        x[t] = xt;
    }
}

hipError_t mh(float*x,float x0, int N, int burn_N, float (*f)(float), int seed) {
    const int samples_per_block = (BLOCK_SIZE * SAMPLES_PER_JOB);
    const int block_num = (N + samples_per_block - 1) / samples_per_block;

    typedef std::chrono::high_resolution_clock Clock;
    
    hipEvent_t start, stop;
    float elapsedTime;
    Clock::time_point t0 = Clock::now();

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }   

    float* dev_x;

    cudaStatus = hipMalloc((void**)&dev_x, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    float xt = x0;
    float std = 1.0;

    std::mt19937 gen(seed);

    std::tuple<float, float> res;
    res = cpu_burn_loop( xt, std, burn_N, cpu_ff, gen);
    //printf("std = %f \n", std::get<1>(res));

    xt = std::get<0>(res);
    std = std::get<1>(res);

    hiprandStateMtgp32* devMTGPStates;
    mtgp32_kernel_params* devKernelParams;

    hipMalloc((void**)&devMTGPStates, block_num * sizeof(hiprandStateMtgp32));

    hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params));

    hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);

    /* Initialize one state per thread block */
    hiprandMakeMTGP32KernelState(devMTGPStates,mtgp32dc_params_fast_11213, devKernelParams, block_num, seed * 117);

    float (*fff)(float);
    hipMemcpyFromSymbol(&fff, HIP_SYMBOL(dev_ff_p), sizeof(f)); //tu nie mogę podać f po prostu tylko muszę ff_gpu_stat


    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

   
    _generate_loop_gpu<<<block_num,BLOCK_SIZE>>>(devMTGPStates, dev_x,N, xt, std, fff);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop)
    );
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(x, dev_x, N * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Clock::time_point t1 = Clock::now();

    auto d = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0);

    printf("Kernel-only time: %f ms\n", elapsedTime);
    printf("All time %d ms \n", d.count());

Error:
    hipFree(dev_x);

    return cudaStatus;
}

extern "C" float* cuda_main(int N, int burn_N)
{
    float* x = (float*)malloc(sizeof(float) * N);


    hipError_t cudaStatus = mh(x, 0.0, N, burn_N, dev_ff_p, 42141241);

   


    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    return x;
}

