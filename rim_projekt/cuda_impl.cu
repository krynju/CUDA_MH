#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <chrono>

#include <stdio.h>
#include <random>
#include <tuple>
#include <math.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hiprand/hiprand_kernel.h>

//#define SAMPLES_PER_THREAD 100000
// Samples per thread should be at most
// 10^3 for 10^6 samples
// 10^4 for 10^7 samples
// 10^5 for 10^8 samples
 
#define BLOCK_SIZE 256
#define BLOCKS_PER_STREAM 1
#define MAX_STREAMS_COUNT 64

// CPU CODE FOR STD TUNING 

float cpu_norm_pdf(float mean, float std, float x) {
    const float a = 1 / sqrt(2 * M_PI);
    return a / std * exp(-0.5 * pow(((x - mean) / std), 2));
}

float cpu_ff(float x) {
    x = x / 100.0;
    return cpu_norm_pdf(0.0, 1.0, x) + cpu_norm_pdf(3.0, 1.0, x) + cpu_norm_pdf(6.0, 1.0, x);
}

std::tuple<float, float> cpu_burn_loop(float xt, float std, int burn_N, float (*f)(float), std::mt19937 gen) {
    float f_xt = f(xt);
    std::uniform_real<float> uni(0, 1);
    const float target = 0.3;
    int accepted = 0;



    for (int t = 0; t < burn_N; t++) {
        std::normal_distribution<> g(xt, std);
        float xc = g(gen);
        float f_xc = f(xc);
        float a = f_xc / f_xt;
        float u = uni(gen);
        if (u <= a) {
            xt = xc;
            f_xt = f_xc;
            accepted += 1;
        }
        float temp = accepted / (t + 1.0);
        float reg = 1 / std * 1000.0 * (temp - target);
        if (!(reg < 0 && abs(reg) > std)) {
            std += reg;
        }
    }
    return std::tuple<float, float>(xt, std);
}


// DEVICE CODE

__device__ float dev_norm_pdf(float mean, float std, float x) {
    const float inv_sqrt_2pi = 0.3989422804014327;
    return inv_sqrt_2pi / std * exp(-pow((x - mean) / std, 2) / 2.0);
}

__device__ float dev_ff(float x) {
    x = x / 100.0;
    return dev_norm_pdf(0.0, 1.0, x) + dev_norm_pdf(3.0, 1.0, x) + dev_norm_pdf(6.0, 1.0, x);
}

__device__ float(*dev_ff_p)(float) = dev_ff;


__global__ void dev_generate(hiprandStateMtgp32* state, float* x, int x_len, float xt, float std,int samples_per_thread, float (*f)(float)) {
    int warp_id = threadIdx.x / 32;
    int id_in_warp = threadIdx.x % 32;
    int warp_range_start = samples_per_thread * BLOCK_SIZE * blockIdx.x + warp_id * 32 * samples_per_thread;
    int warp_range_end = samples_per_thread * BLOCK_SIZE * blockIdx.x + (warp_id + 1) * 32 * samples_per_thread;

    if (warp_range_end > x_len)
        warp_range_end = x_len;

    float f_xt = f(xt);

    // new starting point for each thread
    for (int t = 0; t < 1000; t++) {
        float xc = xt + hiprand_normal(&state[blockIdx.x]) * std / 4.0f ;
        float f_xc = f(xc);
        float a = f_xc / f_xt;
        float u = hiprand_uniform(&state[blockIdx.x]);

        if (u <= a) {
            xt = xc;
            f_xt = f_xc;
        }
    }

    for (int t = warp_range_start; t < warp_range_end  ; t+=32) {
        float xc = xt + hiprand_normal(&state[blockIdx.x])*std ;
        float f_xc = f(xc);
        float a = f_xc / f_xt;
        float u = hiprand_uniform(&state[blockIdx.x]);
  
        if (u <= a) {
            xt = xc;
            f_xt = f_xc;
        }
        x[t + id_in_warp] = xt;
        // W tej pętli każdy wątek w ramach warpu pisze do co 32 adresu w pamięci, żeby lepiej grupować 
        // transfery pamięci
        // TODO: Do porównania z poniższą zakomentowaną wersją, gdzie każdy wątek pisze do swojego przedziału
    }


    // int i = SAMPLES_PER_THREAD * BLOCK_SIZE * blockIdx.x   + threadIdx.x * SAMPLES_PER_THREAD;
    // int sstart_range = i ;
    // int eend_range= i + SAMPLES_PER_THREAD;
    // float f_xt = f(xt);
    // if (eend_range > x_len) { eend_range = x_len; }
    // for (int t = sstart_range; t < eend_range  ; t++) {
    //     float xc = xt + hiprand_normal(&state[blockIdx.x])*std ;
    //     float f_xc = f(xc);
    //     float a = f_xc / f_xt;
    //     float u = hiprand_uniform(&state[blockIdx.x]);
  
    //     if (u <= a) {
    //         xt = xc;
    //         f_xt = f_xc;
    //     }
    //     x[t] = xt;
    // }
}

int mh(float* x, float x0, int N, int burn_N, float (*cpu_f)(float), float (*f)(float), int seed) {
    
   /* int temp = N;
    int p = 0;
    while (temp > 0) {
        temp = temp / 10;
        p++;
    }
    p = pow(10, p - 4);
    p = std::min(p, 100000);
    const int samples_per_thread = std::max(p, 1000);*/
    const int samples_per_thread = 10000;

    const int samples_per_block = (BLOCK_SIZE * samples_per_thread);
    const int block_count = (N + samples_per_block - 1) / samples_per_block;
    const int stream_count = 1 + (block_count - 1) / BLOCKS_PER_STREAM;
    const int samples_per_stream = BLOCKS_PER_STREAM * BLOCK_SIZE * samples_per_thread;


    typedef std::chrono::high_resolution_clock Clock;
    hipEvent_t start, stop;
    float elapsedTime;
    Clock::time_point t0 = Clock::now();

    checkCudaErrors(hipSetDevice(0));

    float* dev_x;

    size_t free = 0, total= 0;
    hipMemGetInfo(&free, &total);

    const int MEMORY_POOLS_AVAILABLE = free / sizeof(float) / samples_per_stream;
     int MEMORY_POOLS = MEMORY_POOLS_AVAILABLE * 8 / 10;

    if (stream_count < MEMORY_POOLS) MEMORY_POOLS = stream_count;

    hipHostRegister(x, N * sizeof(float), 0);


    // Strojenie odchylenia standardowego wspólne na CPU
    float xt = x0;
    float std = 1.0;
    std::mt19937 gen(seed);
    std::tuple<float, float> res;
    res = cpu_burn_loop(xt, std, burn_N, cpu_f, gen);
    //printf("std = %f \n", std::get<1>(res));

    xt = std::get<0>(res);
    std = std::get<1>(res);




    float (*fff)(float);
    checkCudaErrors(hipMemcpyFromSymbol(&fff, HIP_SYMBOL(dev_ff_p), sizeof(f)));
    // TODO: tutaj jest problem, że nie można podać do hipMemcpyFromSymbol na drugi arg
    // funkcji f, która została podana do aktualnej funkcji - coś z tym zrobić trzeba


    int *samples_pool;
    hipHostAlloc((void**)&samples_pool, stream_count * sizeof(int), 0);
    int temp_N = N;
    for (int i = 0; i < stream_count; i++) {
        if (temp_N >= samples_per_stream) {
            samples_pool[i] = samples_per_stream;
            temp_N -= samples_per_stream;
        }
        else {
            samples_pool[i] = temp_N;
        }
    }


   
    checkCudaErrors(hipMalloc((void**)&dev_x, MEMORY_POOLS * samples_per_stream * sizeof(float)));


    hipStream_t *stream;
    hipEvent_t *streamMemoryPoolFreeEvent;

    checkCudaErrors(hipHostAlloc((void**)&stream, stream_count * sizeof(hipStream_t), 0));
    checkCudaErrors(hipHostAlloc((void**)&streamMemoryPoolFreeEvent , stream_count * sizeof(hipEvent_t), 0));
    
    for (int i = 0; i < stream_count; i++) {
        checkCudaErrors(hipStreamCreateWithFlags(stream + i, hipStreamNonBlocking));
        hipEventCreateWithFlags(streamMemoryPoolFreeEvent + i, hipEventDisableTiming & hipEventBlockingSync);
    }

    // Używamy generatora MTGP32, mamy do porównania wersje zaimplementowane na CPU sekwencyjnie i równolegle i
    // one wykorzystują również ten sam generator
    hiprandStateMtgp32* devMTGPStates;
    mtgp32_kernel_params* devKernelParams;

    // Każdy blok dostaje swój generator, każdy generator może wydajnie obsłużyć max 256 wątków w ramach bloku
    checkCudaErrors(hipMalloc((void**)&devMTGPStates, block_count * sizeof(hiprandStateMtgp32)));

    checkCudaErrors(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));

    hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);

    /* Initialize one state per thread block */
    //hiprandMakeMTGP32KernelState(devMTGPStates,mtgp32dc_params_fast_11213, devKernelParams, block_num, seed * 117);
    for (int i = 0; i < block_count; i++) {
        hiprandMakeMTGP32KernelState(devMTGPStates + i, mtgp32dc_params_fast_11213, devKernelParams, 1, seed * 7 * i);
    }
    
   
    for (int s = 0; s < stream_count; s++) {
        //if (s >= MEMORY_POOLS) checkCudaErrors(hipEventSynchronize(streamMemoryPoolFreeEvent[s]));
        if (s >= MEMORY_POOLS) checkCudaErrors(hipStreamWaitEvent(stream[s], streamMemoryPoolFreeEvent[s]));
        dev_generate<<<BLOCKS_PER_STREAM, BLOCK_SIZE, 0, stream[s]>>>(devMTGPStates + s * BLOCKS_PER_STREAM, dev_x + s % MEMORY_POOLS * samples_per_stream, samples_pool[s], xt, std,samples_per_thread, fff);
        checkCudaErrors(hipMemcpyAsync(x + s * samples_per_stream, dev_x + (s % MEMORY_POOLS) * samples_per_stream, samples_pool[s] * sizeof(float), hipMemcpyDeviceToHost, stream[s]));
        if(s+MEMORY_POOLS < stream_count) checkCudaErrors(hipEventRecord(streamMemoryPoolFreeEvent[(s+MEMORY_POOLS)], stream[s]));
    }


    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    for (int s = 0; s < stream_count; s++) {
        checkCudaErrors(hipStreamDestroy(stream[s]));
    }
    Clock::time_point t1 = Clock::now();

    auto d = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0);

    // for kernel time measurement
    // printf("Kernel-only time: %f ms\n", elapsedTime);
    printf("All time %d ms \n", d.count());
 
    checkCudaErrors(hipFree(dev_x));
    hipHostUnregister(x);

    return 0;
}

extern "C" float* cuda_main(int N, int burn_N)
{
    float* x = (float*)malloc(sizeof(float) * N);

    mh(x, 0.0, N, burn_N, cpu_ff,dev_ff_p, 7);

    checkCudaErrors(hipDeviceReset());

    return x;
}

